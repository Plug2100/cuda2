#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include "lodepng.h"
using namespace std;
int help_ = 0;
int smalll = 0;
__global__
void filtr(const unsigned char* dev_input_0, const unsigned char* dev_input_1, const unsigned char* dev_input_2, unsigned char* dev_output_0, unsigned char* dev_output_1, unsigned char* dev_output_2, int width, int height, int cern) {
    //Индекс треда внутри текущего блока
    __shared__  unsigned char r_dev_input_0[36][36];
    __shared__  unsigned char b_dev_input_0[36][36];
    __shared__  unsigned char g_dev_input_0[36][36];

    const unsigned int linearX = (blockIdx.x / 3) * blockDim.x + threadIdx.x;
    const unsigned int linearY = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (linearX >= (width * 3) || linearY >= height) {
        return;
    }

    r_dev_input_0[threadIdx.y+2][threadIdx.x+2] = dev_input_0[(linearY * width + linearX)];
    g_dev_input_0[threadIdx.y+2][threadIdx.x+2] = dev_input_1[(linearY * width + linearX)];
    b_dev_input_0[threadIdx.y+2][threadIdx.x+2] = dev_input_2[(linearY * width + linearX)];
    
    if(threadIdx.x == 31){
        r_dev_input_0[threadIdx.y+2][threadIdx.x+3] = dev_input_0[(linearY * width + linearX+1)];
        g_dev_input_0[threadIdx.y+2][threadIdx.x+3] = dev_input_1[(linearY * width + linearX+1)];
        b_dev_input_0[threadIdx.y+2][threadIdx.x+3] = dev_input_2[(linearY * width + linearX+1)];
    
    }
    if(threadIdx.x == 0 and linearX > 0){
        r_dev_input_0[threadIdx.y+2][threadIdx.x+1] = dev_input_0[(linearY * width + linearX-1)];
        g_dev_input_0[threadIdx.y+2][threadIdx.x+1] = dev_input_1[(linearY * width + linearX-1)];
        b_dev_input_0[threadIdx.y+2][threadIdx.x+1] = dev_input_2[(linearY * width + linearX-1)];
    }
    if(threadIdx.y == 31){
        r_dev_input_0[threadIdx.y+3][threadIdx.x+2] = dev_input_0[((linearY+1) * width + linearX)];
        g_dev_input_0[threadIdx.y+3][threadIdx.x+2] = dev_input_1[((linearY+1) * width + linearX)];
        b_dev_input_0[threadIdx.y+3][threadIdx.x+2] = dev_input_2[((linearY+1) * width + linearX)];
    }
    if(threadIdx.y == 0 and linearY > 0){
        r_dev_input_0[threadIdx.y+1][threadIdx.x+2] = dev_input_0[((linearY-1) * width + linearX)];
        g_dev_input_0[threadIdx.y+1][threadIdx.x+2] = dev_input_1[((linearY-1) * width + linearX)];
        b_dev_input_0[threadIdx.y+1][threadIdx.x+2] = dev_input_2[((linearY-1) * width + linearX)];
    
    }
    if(cern == 1){
        if(threadIdx.x == 31){
        r_dev_input_0[threadIdx.y+2][threadIdx.x+4] = dev_input_0[(linearY * width + linearX+2)];
        g_dev_input_0[threadIdx.y+2][threadIdx.x+4] = dev_input_1[(linearY * width + linearX+2)];
        b_dev_input_0[threadIdx.y+2][threadIdx.x+4] = dev_input_2[(linearY * width + linearX+2)];
    
    }
    if(threadIdx.x == 0 and linearX > 0){
        r_dev_input_0[threadIdx.y+2][threadIdx.x] = dev_input_0[(linearY * width + linearX-2)];
        g_dev_input_0[threadIdx.y+2][threadIdx.x] = dev_input_1[(linearY * width + linearX-2)];
        b_dev_input_0[threadIdx.y+2][threadIdx.x] = dev_input_2[(linearY * width + linearX-2)];
    }
    if(threadIdx.y == 31){
        r_dev_input_0[threadIdx.y+4][threadIdx.x+2] = dev_input_0[((linearY+2) * width + linearX)];
        g_dev_input_0[threadIdx.y+4][threadIdx.x+2] = dev_input_1[((linearY+2) * width + linearX)];
        b_dev_input_0[threadIdx.y+4][threadIdx.x+2] = dev_input_2[((linearY+2) * width + linearX)];
    }
    if(threadIdx.y == 0 and linearY > 0){
        r_dev_input_0[threadIdx.y][threadIdx.x+2] = dev_input_0[((linearY-2) * width + linearX)];
        g_dev_input_0[threadIdx.y][threadIdx.x+2] = dev_input_1[((linearY-2) * width + linearX)];
        b_dev_input_0[threadIdx.y][threadIdx.x+2] = dev_input_2[((linearY-2) * width + linearX)];
    
    }
    }
    __syncthreads();
    if(cern == 1){
        if(linearY > 1 and linearY < (height - 2) and linearX > 1 and linearX < (width - 2)
            or linearY > 1 and linearY < (height - 2) and linearX > (1 + width) and linearX < (2 * width - 2)
            or linearY > 1 and linearY < (height - 2) and linearX > (1 + 2* width) and linearX < (3 * width - 2)) {
            if(blockIdx.x % 3 == 0){
                dev_output_0[(linearY * width + linearX)] = 
                    (r_dev_input_0[threadIdx.y+2][threadIdx.x+2] * 36 +
                    r_dev_input_0[threadIdx.y+2][threadIdx.x+1] * 24 +
                    r_dev_input_0[threadIdx.y+3][threadIdx.x+2] * 24 +
                    r_dev_input_0[threadIdx.y+1][threadIdx.x+2] * 24 +
                    r_dev_input_0[threadIdx.y+2][threadIdx.x+3] * 24 +



                    r_dev_input_0[threadIdx.y+3][threadIdx.x+3] * 16 +
                    r_dev_input_0[threadIdx.y+3][threadIdx.x+1] * 16 +
                    r_dev_input_0[threadIdx.y+1][threadIdx.x+3] * 16 +
                    r_dev_input_0[threadIdx.y+1][threadIdx.x+1] * 16 +
 
                    r_dev_input_0[threadIdx.y+2][threadIdx.x+4] * 6 +
                    r_dev_input_0[threadIdx.y+2][threadIdx.x] * 6 +
                    r_dev_input_0[threadIdx.y+4][threadIdx.x+2] * 6 +
                    r_dev_input_0[threadIdx.y][threadIdx.x+2] * 6 +
 
 
                    r_dev_input_0[threadIdx.y+4][threadIdx.x+4] +
                    r_dev_input_0[threadIdx.y][threadIdx.x+4] +
                    r_dev_input_0[threadIdx.y+4][threadIdx.x] +
                    r_dev_input_0[threadIdx.y][threadIdx.x] +
 
 
                    r_dev_input_0[threadIdx.y][threadIdx.x+1] +
                    r_dev_input_0[threadIdx.y+4][threadIdx.x+1] +
                    r_dev_input_0[threadIdx.y][threadIdx.x+3] +
                    r_dev_input_0[threadIdx.y+4][threadIdx.x+3] +
 
                    r_dev_input_0[threadIdx.y+1][threadIdx.x+4] +
                    r_dev_input_0[threadIdx.y+3][threadIdx.x+4] +
                    r_dev_input_0[threadIdx.y+1][threadIdx.x] +
                    r_dev_input_0[threadIdx.y+3][threadIdx.x]) /  256;
            }
 
 
 
            if(blockIdx.x % 3 == 1){ 
                dev_output_1[(linearY * width + linearX)] = (g_dev_input_0[threadIdx.y+2][threadIdx.x+2] * 36 +
                    g_dev_input_0[threadIdx.y+2][threadIdx.x+1] * 24 +
                    g_dev_input_0[threadIdx.y+3][threadIdx.x+2] * 24 +
                    g_dev_input_0[threadIdx.y+1][threadIdx.x+2] * 24 +
                    g_dev_input_0[threadIdx.y+2][threadIdx.x+3] * 24 +



                    g_dev_input_0[threadIdx.y+3][threadIdx.x+3] * 16 +
                    g_dev_input_0[threadIdx.y+3][threadIdx.x+1] * 16 +
                    g_dev_input_0[threadIdx.y+1][threadIdx.x+3] * 16 +
                    g_dev_input_0[threadIdx.y+1][threadIdx.x+1] * 16 +
 
                    g_dev_input_0[threadIdx.y+2][threadIdx.x+4] * 6 +
                    g_dev_input_0[threadIdx.y+2][threadIdx.x] * 6 +
                    g_dev_input_0[threadIdx.y+4][threadIdx.x+2] * 6 +
                    g_dev_input_0[threadIdx.y][threadIdx.x+2] * 6 +
 
 
                    g_dev_input_0[threadIdx.y+4][threadIdx.x+4] +
                    g_dev_input_0[threadIdx.y][threadIdx.x+4] +
                    g_dev_input_0[threadIdx.y+4][threadIdx.x] +
                    g_dev_input_0[threadIdx.y][threadIdx.x] +
 
 
                    g_dev_input_0[threadIdx.y][threadIdx.x+1] +
                    g_dev_input_0[threadIdx.y+4][threadIdx.x+1] +
                    g_dev_input_0[threadIdx.y][threadIdx.x+3] +
                    g_dev_input_0[threadIdx.y+4][threadIdx.x+3] +
 
                    g_dev_input_0[threadIdx.y+1][threadIdx.x+4] +
                    g_dev_input_0[threadIdx.y+3][threadIdx.x+4] +
                    g_dev_input_0[threadIdx.y+1][threadIdx.x] +
                    g_dev_input_0[threadIdx.y+3][threadIdx.x]) /  256;
            }
 
 
            if(blockIdx.x % 3 == 2){ 
                dev_output_2[(linearY * width + linearX)] = (b_dev_input_0[threadIdx.y+2][threadIdx.x+2] * 36 +
                    b_dev_input_0[threadIdx.y+2][threadIdx.x+1] * 24 +
                    b_dev_input_0[threadIdx.y+3][threadIdx.x+2] * 24 +
                    b_dev_input_0[threadIdx.y+1][threadIdx.x+2] * 24 +
                    b_dev_input_0[threadIdx.y+2][threadIdx.x+3] * 24 +



                    b_dev_input_0[threadIdx.y+3][threadIdx.x+3] * 16 +
                    b_dev_input_0[threadIdx.y+3][threadIdx.x+1] * 16 +
                    b_dev_input_0[threadIdx.y+1][threadIdx.x+3] * 16 +
                    b_dev_input_0[threadIdx.y+1][threadIdx.x+1] * 16 +
 
                    b_dev_input_0[threadIdx.y+2][threadIdx.x+4] * 6 +
                    b_dev_input_0[threadIdx.y+2][threadIdx.x] * 6 +
                    b_dev_input_0[threadIdx.y+4][threadIdx.x+2] * 6 +
                    b_dev_input_0[threadIdx.y][threadIdx.x+2] * 6 +
 
 
                    b_dev_input_0[threadIdx.y+4][threadIdx.x+4] +
                    b_dev_input_0[threadIdx.y][threadIdx.x+4] +
                    b_dev_input_0[threadIdx.y+4][threadIdx.x] +
                    b_dev_input_0[threadIdx.y][threadIdx.x] +
 
 
                    b_dev_input_0[threadIdx.y][threadIdx.x+1] +
                    b_dev_input_0[threadIdx.y+4][threadIdx.x+1] +
                    b_dev_input_0[threadIdx.y][threadIdx.x+3] +
                    b_dev_input_0[threadIdx.y+4][threadIdx.x+3] +
 
                    b_dev_input_0[threadIdx.y+1][threadIdx.x+4] +
                    b_dev_input_0[threadIdx.y+3][threadIdx.x+4] +
                    b_dev_input_0[threadIdx.y+1][threadIdx.x] +
                    b_dev_input_0[threadIdx.y+3][threadIdx.x]) /  256;
            }
        }
    }
    else if(cern == 2){
        if(linearY > 0 and linearY < (height - 1) and linearX > 0 and linearX < (width - 1)
            or linearY > 0 and linearY < (height - 1) and linearX > width and linearX < (2*width - 1)
            or linearY > 0 and linearY < (height - 1) and linearX > 2*width and linearX < (3*width - 1)) {
            if(blockIdx.x % 3 == 0){ 
                dev_output_0[(linearY * width + linearX)] =
                    (r_dev_input_0[threadIdx.y+2][threadIdx.x+2] +
                          r_dev_input_0[threadIdx.y+1][threadIdx.x+3] +
                          r_dev_input_0[threadIdx.y+2][threadIdx.x+3] +
                          r_dev_input_0[threadIdx.y+3][threadIdx.x+3] +
                          r_dev_input_0[threadIdx.y+1][threadIdx.x+2] +
                          r_dev_input_0[threadIdx.y+3][threadIdx.x+2] +
                          r_dev_input_0[threadIdx.y+1][threadIdx.x+1] +
                          r_dev_input_0[threadIdx.y+2][threadIdx.x+1] +
                          r_dev_input_0[threadIdx.y+2][threadIdx.x+2]) / 9;
            }
            if(blockIdx.x % 3 == 1){ 
                dev_output_1[(linearY * width + linearX)] =
                    (g_dev_input_0[threadIdx.y+2][threadIdx.x+2] +
                          g_dev_input_0[threadIdx.y+1][threadIdx.x+3] +
                          g_dev_input_0[threadIdx.y+2][threadIdx.x+3] +
                          g_dev_input_0[threadIdx.y+3][threadIdx.x+3] +
                          g_dev_input_0[threadIdx.y+1][threadIdx.x+2] +
                          g_dev_input_0[threadIdx.y+3][threadIdx.x+2] +
                          g_dev_input_0[threadIdx.y+1][threadIdx.x+1] +
                          g_dev_input_0[threadIdx.y+2][threadIdx.x+1] +
                          g_dev_input_0[threadIdx.y+2][threadIdx.x+2]) / 9;
            }
            if(blockIdx.x % 3 == 2){ 
                dev_output_2[(linearY * width + linearX)] =
                    (b_dev_input_0[threadIdx.y+2][threadIdx.x+2] +
                          b_dev_input_0[threadIdx.y+1][threadIdx.x+3] +
                          b_dev_input_0[threadIdx.y+2][threadIdx.x+3] +
                          b_dev_input_0[threadIdx.y+3][threadIdx.x+3] +
                          b_dev_input_0[threadIdx.y+1][threadIdx.x+2] +
                          b_dev_input_0[threadIdx.y+3][threadIdx.x+2] +
                          b_dev_input_0[threadIdx.y+1][threadIdx.x+1] +
                          b_dev_input_0[threadIdx.y+2][threadIdx.x+1] +
                          b_dev_input_0[threadIdx.y+2][threadIdx.x+2]) / 9;
            }
        }
    }
    else if(cern == 3){
        if(linearY > 0 and linearY < (height - 1) and linearX > 0 and linearX < (width - 1)
            or linearY > 0 and linearY < (height - 1) and linearX > width and linearX < (2*width - 1)
            or linearY > 0 and linearY < (height - 1) and linearX > 2*width and linearX < (3*width - 1)){
            if(blockIdx.x % 3 == 0){    
                dev_output_0[(linearY * width + linearX)] =
                    r_dev_input_0[threadIdx.y+2][threadIdx.x+2] * 8 -
                          r_dev_input_0[threadIdx.y+1][threadIdx.x+3] -
                          r_dev_input_0[threadIdx.y+2][threadIdx.x+3] -
                          r_dev_input_0[threadIdx.y+3][threadIdx.x+3] -
                          r_dev_input_0[threadIdx.y+1][threadIdx.x+2] -
                          r_dev_input_0[threadIdx.y+3][threadIdx.x+2] -
                          r_dev_input_0[threadIdx.y+1][threadIdx.x+1] -
                          r_dev_input_0[threadIdx.y+2][threadIdx.x+1] -
                          r_dev_input_0[threadIdx.y+2][threadIdx.x+2];
            }
            if(blockIdx.x % 3 == 1){ 
                dev_output_1[(linearY * width + linearX)]=
                    g_dev_input_0[threadIdx.y+2][threadIdx.x+2] * 8 -
                          g_dev_input_0[threadIdx.y+1][threadIdx.x+3] -
                          g_dev_input_0[threadIdx.y+2][threadIdx.x+3] -
                          g_dev_input_0[threadIdx.y+3][threadIdx.x+3] -
                          g_dev_input_0[threadIdx.y+1][threadIdx.x+2] -
                          g_dev_input_0[threadIdx.y+3][threadIdx.x+2] -
                          g_dev_input_0[threadIdx.y+1][threadIdx.x+1] -
                          g_dev_input_0[threadIdx.y+2][threadIdx.x+1] -
                          g_dev_input_0[threadIdx.y+2][threadIdx.x+2];
            }
            if(blockIdx.x % 3 == 2){ 
                dev_output_2[(linearY * width + linearX)] =
                    b_dev_input_0[threadIdx.y+2][threadIdx.x+2] * 8-
                          b_dev_input_0[threadIdx.y+1][threadIdx.x+3] -
                          b_dev_input_0[threadIdx.y+2][threadIdx.x+3] -
                          b_dev_input_0[threadIdx.y+3][threadIdx.x+3] -
                          b_dev_input_0[threadIdx.y+1][threadIdx.x+2] -
                          b_dev_input_0[threadIdx.y+3][threadIdx.x+2] -
                          b_dev_input_0[threadIdx.y+1][threadIdx.x+1] -
                          b_dev_input_0[threadIdx.y+2][threadIdx.x+1] -
                          b_dev_input_0[threadIdx.y+2][threadIdx.x+2];
            }
        }
    }
}
 
 
 
void funk(const char* input_file, const char* output_file, int filtr_n){
    vector<unsigned char> in_image;
    unsigned int width, height;
    // Load the data
    unsigned error = lodepng::decode(in_image, width, height, input_file);
    float all_all = 0;
    float caunting_all = 0;
    hipEvent_t all_start;
    hipEvent_t all_stop;
    hipEvent_t start_caunting;
    hipEvent_t stop_caunting;
    hipEventCreate(&all_start);
    hipEventCreate(&all_stop);
    hipEventCreate(&start_caunting);
    hipEventCreate(&stop_caunting);
 
 
    // Prepare the data
    unsigned char* input_image_0 = new unsigned char[(in_image.size())/4];
    unsigned char* input_image_1 = new unsigned char[(in_image.size())/4];
    unsigned char* input_image_2 = new unsigned char[(in_image.size())/4];
 
    unsigned char* output_image_0 = new unsigned char[(in_image.size())/4];
    unsigned char* output_image_1 = new unsigned char[(in_image.size())/4];
    unsigned char* output_image_2 = new unsigned char[(in_image.size())/4];
 
    int where_0 = 0;
    int where_1 = 0;
    int where_2 = 0;
    for(int i = 0; i < in_image.size(); ++i) {
        if(i % 4 == 0){
            input_image_0[where_0] = in_image.at(i);
            where_0++;
        }
        if(i % 4 == 1){
            input_image_1[where_1] = in_image.at(i);
            where_1++;
        }
        if(i % 4 == 2){
            input_image_2[where_2] = in_image.at(i);
            where_2++;
        }
    }
 
    hipEventRecord(all_start);
    hipEventSynchronize(all_start);
 
 
    unsigned char* dev_input_0;
    unsigned char* dev_input_1;
    unsigned char* dev_input_2;

    unsigned char* dev_output_0;
    unsigned char* dev_output_1;
    unsigned char* dev_output_2;
    if(help_ == 0){
        help_ = 1;
        hipMalloc( (void**) &dev_input_0, width*height*sizeof(unsigned char));
        hipMalloc( (void**) &dev_input_1, width*height*sizeof(unsigned char));
        hipMalloc( (void**) &dev_input_2, width*height*sizeof(unsigned char));
        hipMalloc( (void**) &dev_output_0, width*height*sizeof(unsigned char));
        hipMalloc( (void**) &dev_output_1, width*height*sizeof(unsigned char));
        hipMalloc( (void**) &dev_output_2, width*height*sizeof(unsigned char));
    }
    hipMemcpy( dev_input_0, input_image_0, width*height*sizeof(unsigned char), hipMemcpyHostToDevice ); 
    hipMemcpy( dev_input_1, input_image_1, width*height*sizeof(unsigned char), hipMemcpyHostToDevice );
    hipMemcpy( dev_input_2, input_image_2, width*height*sizeof(unsigned char), hipMemcpyHostToDevice );
 
 
 
 
   
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int help = floor(sqrt(prop.maxThreadsPerBlock));
   // help = 61
    if(help > height){
        help = height;
    }
    if(help > width){
        help = width;
    }  
    dim3 blockDims(help, help,1);
    dim3 gridDims(ceil(width / help) * 3, ceil(height / help), 1 );
 
 
 
    hipEventRecord(start_caunting);
    hipEventSynchronize(start_caunting);
 
    filtr<<<gridDims, blockDims>>>(dev_input_0, dev_input_1, dev_input_2, dev_output_0, dev_output_1, dev_output_2, width, height, filtr_n);
    hipEventRecord(stop_caunting);
    hipEventSynchronize(stop_caunting);
    hipEventElapsedTime(&caunting_all, start_caunting, stop_caunting);
 
    hipMemcpy(output_image_0, dev_output_0, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost );
    hipMemcpy(output_image_1, dev_output_1, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost );
    hipMemcpy(output_image_2, dev_output_2, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost );
 

 
    hipEventRecord(all_stop);
    hipEventSynchronize(all_stop);
    hipEventElapsedTime(&all_all, all_start, all_stop);
    where_0 = 0;
    where_1 = 0;
    where_2 = 0;
    vector<unsigned char> out_image;
    for(int i = 0; i < in_image.size(); ++i) {
        if(i % 4 == 0){
            out_image.push_back(output_image_0[where_0]);
            where_0++;
        }
        if(i % 4 == 1){
            out_image.push_back(output_image_1[where_1]);
            where_1++;
        }
        if(i % 4 == 2){
            out_image.push_back(output_image_2[where_2]);
            where_2++;
        }
        if(i % 4 == 3){
            out_image.push_back(255);
        }
    }
 
    error = lodepng::encode(output_file, out_image, width, height);
 
 
   // cout <<"caunting: " <<caunting_all << endl;
   // cout << "all time "<< all_all << endl;
}
 
 




















void funks(const char* input_file, const char* input_file2, const char* input_file3, const char* output_file, int filtr_n){
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    vector<unsigned char> in_image;
    vector<unsigned char> in_image2;
    vector<unsigned char> in_image3;

    unsigned int width, height;
    // Load the data
    unsigned error = lodepng::decode(in_image, width, height, input_file);
    error = lodepng::decode(in_image2, width, height, input_file2);
    error = lodepng::decode(in_image3, width, height, input_file3);


    float all_all = 0;
    float caunting_all = 0;
    hipEvent_t all_start;
    hipEvent_t all_stop;
    hipEvent_t start_caunting;
    hipEvent_t stop_caunting;
    hipEventCreate(&all_start);
    hipEventCreate(&all_stop);
    hipEventCreate(&start_caunting);
    hipEventCreate(&stop_caunting);
 
 
    // Prepare the data
    
    unsigned char* input_image_0 = new unsigned char[3*(in_image.size())/4];
    unsigned char* input_image_1 = new unsigned char[3*(in_image.size())/4];
    unsigned char* input_image_2 = new unsigned char[3*(in_image.size())/4];
 
    unsigned char* output_image_0 = new unsigned char[3*(in_image.size())/4];
    unsigned char* output_image_1 = new unsigned char[3*(in_image.size())/4];
    unsigned char* output_image_2 = new unsigned char[3*(in_image.size())/4];
    
    int where_0 = 0;
    int where_1 = 0;
    int where_2 = 0;

    
    for(int i = 0; i < in_image.size(); ++i) {
        if(i % 4 == 0){
            input_image_0[where_0] = in_image.at(i);
            where_0++;
        }
        if(i % 4 == 1){
            input_image_1[where_1] = in_image.at(i);
            where_1++;
        }
        if(i % 4 == 2){
            input_image_2[where_2] = in_image.at(i);
            where_2++;
        }
    }
    for(int i = 0; i < in_image2.size(); ++i) {
        if(i % 4 == 0){
            input_image_0[where_0] = in_image2.at(i);
            where_0++;
        }
        if(i % 4 == 1){
            input_image_1[where_1] = in_image2.at(i);
            where_1++;
        }
        if(i % 4 == 2){
            input_image_2[where_2] = in_image2.at(i);
            where_2++;
        }
    }
    for(int i = 0; i < in_image3.size(); ++i) {
        if(i % 4 == 0){
            input_image_0[where_0] = in_image3.at(i);
            where_0++;
        }
        if(i % 4 == 1){
            input_image_1[where_1] = in_image3.at(i);
            where_1++;
        }
        if(i % 4 == 2){
            input_image_2[where_2] = in_image3.at(i);
            where_2++;
        }
    }
 
    hipEventRecord(all_start);
    hipEventSynchronize(all_start);
 
 
    unsigned char* dev_input_0;
    unsigned char* dev_input_1;
    unsigned char* dev_input_2;

    unsigned char* dev_output_0;
    unsigned char* dev_output_1;
    unsigned char* dev_output_2;

    width = width * 3;
    hipMalloc( (void**) &dev_input_0, width*height*sizeof(unsigned char));
    hipMalloc( (void**) &dev_input_1, width*height*sizeof(unsigned char));
    hipMalloc( (void**) &dev_input_2, width*height*sizeof(unsigned char));
    hipMalloc( (void**) &dev_output_0, width*height*sizeof(unsigned char));
    hipMalloc( (void**) &dev_output_1, width*height*sizeof(unsigned char));
    hipMalloc( (void**) &dev_output_2, width*height*sizeof(unsigned char));
    hipMemcpyAsync( dev_input_0, input_image_0, width*height*sizeof(unsigned char), hipMemcpyHostToDevice, stream ); 
    hipMemcpyAsync( dev_input_1, input_image_1, width*height*sizeof(unsigned char), hipMemcpyHostToDevice, stream );
    hipMemcpyAsync( dev_input_2, input_image_2, width*height*sizeof(unsigned char), hipMemcpyHostToDevice, stream );




   
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int help = floor(sqrt(prop.maxThreadsPerBlock));
    if(help > height){
        help = height;
    }
    if(help > width){
        help = width;
    }  
    dim3 blockDims(help, help,1);
    dim3 gridDims(ceil(width / help) * 9 , ceil(height / help), 1 );
 
 
 
    hipEventRecord(start_caunting);
    hipEventSynchronize(start_caunting);

    filtr<<<gridDims, blockDims, 0, stream>>>(dev_input_0, dev_input_1, dev_input_2, dev_output_0, dev_output_1, dev_output_2, width, height, filtr_n);

    hipDeviceSynchronize();
    hipEventRecord(stop_caunting);
    hipEventSynchronize(stop_caunting);
    hipEventElapsedTime(&caunting_all, start_caunting, stop_caunting);
    
    hipMemcpyAsync(output_image_0, dev_output_0, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost , stream);
    hipMemcpyAsync(output_image_1, dev_output_1, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost , stream);
    hipMemcpyAsync(output_image_2, dev_output_2, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost , stream);   
    

 
    hipEventRecord(all_stop);
    hipEventSynchronize(all_stop);
    hipEventElapsedTime(&all_all, all_start, all_stop);
    where_0 = 0;
    where_1 = 0;
    where_2 = 0;
    vector<unsigned char> out_image;
    vector<unsigned char> out_image2;
    vector<unsigned char> out_image3;

    for(int i = 0; i < in_image.size(); ++i) {
        if(i % 4 == 0){
            out_image.push_back(output_image_0[where_0]);
            where_0++;
        }
        if(i % 4 == 1){
            out_image.push_back(output_image_1[where_1]);
            where_1++;
        }
        if(i % 4 == 2){
            out_image.push_back(output_image_2[where_2]);
            where_2++;
        }
        if(i % 4 == 3){
            out_image.push_back(255);
        }
    }
    for(int i = 0; i < in_image.size(); ++i) {
        if(i % 4 == 0){
            out_image2.push_back(output_image_0[where_0]);
            where_0++;
        }
        if(i % 4 == 1){
            out_image2.push_back(output_image_1[where_1]);
            where_1++;
        }
        if(i % 4 == 2){
            out_image2.push_back(output_image_2[where_2]);
            where_2++;
        }
        if(i % 4 == 3){
            out_image2.push_back(255);
        }
    }
    for(int i = 0; i < in_image.size(); ++i) {
        if(i % 4 == 0){
            out_image3.push_back(output_image_0[where_0]);
            where_0++;
        }
        if(i % 4 == 1){
            out_image3.push_back(output_image_1[where_1]);
            where_1++;
        }
        if(i % 4 == 2){
            out_image3.push_back(output_image_2[where_2]);
            where_2++;
        }
        if(i % 4 == 3){
            out_image3.push_back(255);
        }
    }
    const char* output_file2 = "2_out.png";
    const char* output_file3 = "3_out.png";

    error = lodepng::encode(output_file, out_image, width/3, height);
    error = lodepng::encode(output_file2, out_image2, width/3, height);
    error = lodepng::encode(output_file3, out_image3, width/3, height);
 //   cout <<"caunting: " <<caunting_all << endl;
   // cout << "all time "<< all_all << endl; 
}
 
 
 
 
 
 
 
 
int main(int argc, char** argv) {
    string cern = argv[1];
    string size = argv[2];
    const char* input_file;
    const char* output_file;
    // Read the arguments
    int filtr_n = 0;
    if(cern == "blur5"){
        filtr_n = 1;
    }
    else if(cern == "blur"){
        filtr_n = 2;
    }
    else if(cern == "edge_detection"){
        filtr_n = 3;
    }
    if(size == "small"){
        for(int i = 0; i < 1; i++){
            smalll = 1; 
            const char* input_file2 = "2.png";        
            const char* input_file3 = "3.png";
            input_file = "1.png";
            output_file = "1_out.png";
            funks(input_file,input_file2, input_file3, output_file, filtr_n);
        }
    }
    else{
        input_file = "big.png";
        output_file = "big_out.png";
        funk(input_file, output_file, filtr_n);
    }
 
    return 0;
}